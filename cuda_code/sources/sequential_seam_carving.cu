#include "hip/hip_runtime.h"
#include "./utils.cu"
#include <hip/hip_runtime.h>

/****************************************************************************/
/* IMPLEMENTATION OF SEQUENTIAL SEAM CARVING */
/****************************************************************************/
struct pair_int_int {
    	int first;
    	int second;
};

int d[3] = {-1,0,1};

void convert_rgb_to_grayscale(uchar3 * inPixels, int width, int height, uint8_t * outPixels)
{
	// Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
	for (int r = 0; r < height; r++)
	{
		for (int c = 0; c < width; c++)
		{
			int i = r * width + c;
			uint8_t red = inPixels[i].x;
			uint8_t green = inPixels[i].y;
			uint8_t blue = inPixels[i].z;
			outPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
		}
	}
	
}

void apply_filter_(uint8_t* inPixels, int width, int height, float * filter, int filterWidth, uint8_t* outPixels) {
	int half_fWidth = filterWidth / 2;
	// Loop over image
	for (int r = 0; r < height; ++r) {
		for (int c = 0; c < width; ++c) {
			// Set output element to default value
			int pos = r * width + c;
			float fpixel = 0; //Use float to avoiding rounding error during summation

			// Loop over filter
			for (int f_r = -half_fWidth; f_r <= half_fWidth; ++f_r) {
				for (int f_c = -half_fWidth; f_c <= half_fWidth; ++f_c) {
					// Get the matrix element corresponding to the filter element's position
					int i = r + f_r,
						j = c + f_c;
					// Clamp the row and column indices if they are out of bounds
					i = i < 0 ? 0 : i;
					i = i > height - 1 ? height - 1 : i;
					j = j < 0 ? 0 : j;
					j = j > width - 1 ? width - 1 : j;
					// Calculate input position and filter position in 1D
					int in_pos = i * width + j,
						f_pos = (f_r + half_fWidth) * filterWidth + (f_c + half_fWidth);
					// Do convolution
					fpixel += inPixels[in_pos] * filter[f_pos];
				}
			}

			outPixels[pos] = (uint8_t) fpixel;
		}
	}
}

// void apply_filter(uint8_t * inPixels, int width, int height, float * filter, int filterWidth, int * outPixels)
// {
// 	for (int r = 0; r < height; r++) {
//             for (int c = 0; c < width; c++) {
// 				// filter with convolution
// 				int count = 0;
// 				float res = 0;
//                 for (int r_filter = -filterWidth / 2; r_filter <= filterWidth/2; r_filter++){
// 					for (int c_filter = -filterWidth / 2; c_filter <= filterWidth/2; c_filter++) {
// 						int r_new = min(max(r + r_filter, 0),height-1);
// 						int c_new = min(max(c + c_filter, 0),width-1);
// 						int index = r_new * width + c_new;
// 						res += filter[count] * (float)inPixels[index];
// 						count++;
// 					}
// 				}
// 				int i = r * width + c;
// 				outPixels[i] = res;
//             }
//         }
// }

void calc_px_importance(uint8_t *inPixels_1 , uint8_t *inPixels_2, int* outPixels,int width, int height)
{
	for (int i = 0; i < height*width; i++) 
		outPixels[i] = abs(inPixels_1[i])  + abs(inPixels_2[i]);	
}

void create_important_matrix(int * inPixels ,int width, int height, 
			int * outMatrix, int * outMatrixTrace)
{
	for (int r = 0; r < height; r++) 
        for (int c = 0; c < width; c++){ 
			outMatrix[r*width + c] = 1000000000;
			for (int k = 0; k < 3; k++)
				if (r > 0){
					int tmp = outMatrix[(r-1)*width + c+d[k]] + inPixels[r*width + c];
					if (0 <= c+d[k] && c+d[k] < width && 
						outMatrix[r*width + c] > tmp){
						outMatrix[r*width + c] = tmp;
						outMatrixTrace[r*width + c] = k;
					}
				}
				else
				{
					outMatrix[r*width + c] = inPixels[r*width + c];
					outMatrixTrace[r*width + c] = -1;
				}
		}
}

int compare(const void *a, const void *b) {
  
    pair_int_int *pairA = (pair_int_int *)a;
    pair_int_int *pairB = (pair_int_int *)b;
  
    return pairA->first > pairB->first;
}

int compare_position(const void *a, const void *b) {
  
    pair_int_int *pairA = (pair_int_int *)a;
    pair_int_int *pairB = (pair_int_int *)b;
  
    return pairA->first == pairB->first ? pairA->second < pairB->second : pairA->first < pairA->first;
}

int get_trace(int *important_matrix_trace, int position,int width, int height, pair_int_int *res)
{
	int tmp_height = height, tmp_position = position;
	int tmp_position_old = position;

	while (tmp_height--){
		printf("x%i ", tmp_height);
		int count = 0;
		if (tmp_height==0) break;
		while (important_matrix_trace[tmp_height*width+tmp_position] == -1){
			if (count == 3) return 0;
			tmp_position = tmp_position_old + d[count];
			count += 1;
		}
		res[tmp_height] = {tmp_height, tmp_position};
		tmp_position_old = tmp_position;
		int tmp = d[important_matrix_trace[tmp_height*width+tmp_position]];
		important_matrix_trace[tmp_height*width+tmp_position] = -1;
		tmp_position += tmp;
	}
	res[tmp_height] = {tmp_height, tmp_position};
	return 1;
}

int get_k_best(int * important_matrix, int * important_matrix_trace, 
				int width,int height, int k, pair_int_int * k_best)
{
	pair_int_int * tmp_list = (pair_int_int *)malloc(width *sizeof(pair_int_int));
	for (int i=0; i < width; i++)
	{
		tmp_list[i].first = important_matrix[(height-1) * width + i];
		tmp_list[i].second = i;
	}
	qsort(tmp_list, width, sizeof(pair_int_int),compare);

	int count = 0;
	for (int i=0; i<width && count<k; i++){
		// get trace không thể song song
		count += get_trace(important_matrix_trace,tmp_list[i].second,width, height,k_best+count*height);
		printf("%i ", count);
	}
	return count;
}

__global__ void dp_cuda(int * inPixels ,int width, int height, int r, 
			int * outMatrix, int * outMatrixTrace)
{
	int d[3] = {-1,0,1};
	
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	outMatrix[r*width + c] = 1000000000;
	for (int k = 0; k < 3; k++)
		if (r > 0){
			int tmp = outMatrix[(r-1)*width + c+d[k]] + inPixels[r*width + c];
			if (0 <= c+d[k] && c+d[k] < width && 
				outMatrix[r*width + c] > tmp){
				outMatrix[r*width + c] = tmp;
				outMatrixTrace[r*width + c] = k;
			}
		}
		else
		{
			outMatrix[r*width + c] = inPixels[r*width + c];
			outMatrixTrace[r*width + c] = -1;
		}
}

__global__ void create_pair(int * d_in, int width, int height, pair_int_int * out_pair){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<width){
		out_pair[i].first = d_in[i];
		out_pair[i].second = i;
	}
}

int get_k_best_cuda(int * important_matrix, int * important_matrix_trace, 
				int width,int height, int k, pair_int_int * k_best,int blockSize)
{
	size_t pair_nBytes = width * sizeof(pair_int_int);
  	size_t nBytes = width * sizeof(int);
	pair_int_int * tmp_list = (pair_int_int *)malloc(width *sizeof(pair_int_int));
	dim3 gridSize_x1((width - 1) / blockSize + 1);
	int * d_important_matrix;
	pair_int_int * out_pair;
	CHECK(hipMalloc(&d_important_matrix, nBytes));
	CHECK(hipMalloc(&out_pair, pair_nBytes));
	int index = (height-1)*width;
	CHECK(hipMemcpy(d_important_matrix, important_matrix+index, nBytes, hipMemcpyHostToDevice));
	create_pair<<<gridSize_x1, blockSize>>>(d_important_matrix,width, height, out_pair);
	hipDeviceSynchronize();
    CHECK(hipGetLastError());
	CHECK(hipMemcpy(tmp_list, out_pair, pair_nBytes, hipMemcpyDeviceToHost));

	// for (int i=0; i < width; i++)
	// {
	// 	tmp_list[i].first = important_matrix[(height-1) * width + i];
	// 	tmp_list[i].second = i;
	// }
	
	qsort(tmp_list, width, sizeof(pair_int_int),compare);

	// số lượng K quá nhỏ để nên làm song song
	int count = 0;
	for (int i=0; i<width && count<k; i++){
		// get trace không thể song song
		count += get_trace(important_matrix_trace,tmp_list[i].second,width, height,k_best+count*height);
		// printf("%i ", count);
	}
	return count;
}

void create_important_matrix_cuda(int * important_pixels ,int width, int height, 
			int * outMatrix, int * outMatrixTrace, int blockSize){
	// nice version
	size_t nBytes = width * height * sizeof(int);
	dim3 gridSize_x1((width - 1) / blockSize + 1);
	
	int * d_important_pixels, * d_important_matrix, * d_important_matrix_trace;
	CHECK(hipMalloc(&d_important_pixels, nBytes));
	CHECK(hipMalloc(&d_important_matrix, nBytes));
	CHECK(hipMalloc(&d_important_matrix_trace, nBytes));
	CHECK(hipMemcpy(d_important_pixels, important_pixels, nBytes, hipMemcpyHostToDevice));
	for (int r=0; r<height; r++){
		dp_cuda<<<gridSize_x1, blockSize>>>(d_important_pixels,width, height, r, d_important_matrix,d_important_matrix_trace);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
	}
	CHECK(hipMemcpy(outMatrix, d_important_matrix, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(outMatrixTrace, d_important_matrix_trace, nBytes, hipMemcpyDeviceToHost));	
}

void applyKSeams(uchar3* inPixels, uchar3* outPixels, int width, int height, pair_int_int* seams, int k, int mode) {
	if (mode == 0) {
		// Reduce image size
		// Loop for each row of input image
		for (int i = 0; i < height; ++i) {
			// Use 2 pointers to remove seams in a row
			int outIte = 0, inIte = 0, seamIte = 0;
			while (inIte < width) {
				if (seamIte >= k || inIte != seams[seamIte * height + i].second) {
					outPixels[outIte] = inPixels[inIte];
					++outIte;
				} else {
					inIte++;
					seamIte++;
				}
			}
		}
	} else {
		// Enlarge image size
		for (int i = 0; i < height; ++i) {
			// Use 2 pointers to duplicate seams in a row
			int outIte = width + k - 1, inIte = width - 1, seamIte = k - 1;
			while (inIte >= 0) {
				outPixels[outIte] = inPixels[inIte];
				--outIte;
				if (seamIte >= 0 && inIte == seams[seamIte * height + i].second) {
					outPixels[outIte] = inPixels[inIte];
					--outIte;
				}
				--inIte;
			}
		}
	}
	// // Copy the applied pixels to output
	// for (int i = 0 ; i < height; ++i) {
	// 	int row = i * desiredWidth;
	// 	for (int j = 0; j < desiredWidth; ++j) {
	// 		int index = row + j;
	// 		outPixels[index] = inPixels[index];
	// 	}
	// }
}

int main(int argc, char ** argv) {
    // Parse command-line arguments
    if (argc != 4 && argc != 6)
	{
		printf("Invalid run arguments.\nCommand: <executable> <path-to-input-PNM-image> <path-to-output-PNM-image> <desired-image-width> <max-seam-ratio> <cuda-block-size>\n");
		return EXIT_FAILURE;
	}
    char* inImg = argv[1];
    char* outImg = argv[2];
	int desiredWidth = atoi(argv[3]);
    int blockSize = 32;
	float maxSeamRatio = 0.5;
    if (argc >= 5)
        maxSeamRatio = atof(argv[4]);
	if (argc == 6)
		blockSize = atoi(argv[5]);
    printf("Run with block size: %d x %d - Max seam ratio: %.2f\n", blockSize, blockSize, maxSeamRatio);

    // Read input image
    int numChannels, width, height;
	uchar3 *inPixels;
	readPnm(inImg, numChannels, width, height, inPixels);
	if (numChannels != 3)
		return EXIT_FAILURE; // Input image must be RGB
	printf("Image size (width x height): %i x %i\n\n", width, height);
	printf("Desired image width: %d\n", desiredWidth);

	// Calculate number of seams needed
	int k = 0;
	int leftover = 0;
	int mode = 0; // Mode: 0 = reduce size, 1 = enlarge size
	if (desiredWidth < width) {
		k = width - desiredWidth;
	} else {
		k = desiredWidth - width;
		if (k > int(width / 2)) {
			leftover = k - int(width / 2);
			k = int(width / 2);
		}
		mode = 1;
	}

	// A variable to keep total run time
	float total_time_sequential = 0;

	// Convert RGB image to grayscale for easy processing
	uint8_t *grayscalePixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
	GpuTimer timer;
	
	timer.Start();
	convert_rgb_to_grayscale(inPixels, width, height, grayscalePixels);
	timer.Stop();
	float time = timer.Elapsed();
	printf("Convert RGB to Grayscale - Processing time: %f ms\n\n", time);
	total_time_sequential += time;

	// Do convolution with edge detection filters
	float filter1[9] = {1,0,-1,2,0,-2,1,0,-1}; // x-Sobel filter
	float filter2[9] = {1,2,1,0,0,0,-1,-2,-1}; // y-Sobel filter
	int filterWidth = 3;
	uint8_t * filteredPixels_1 = (uint8_t *)malloc(width * height * sizeof(uint8_t));
	uint8_t * filteredPixels_2 = (uint8_t *)malloc(width * height * sizeof(uint8_t));
	
	timer.Start();
	apply_filter_(grayscalePixels, width, height, filter1, filterWidth, filteredPixels_1);
	timer.Stop();
	time = timer.Elapsed();
	printf("Apply x-Sobel filter - Processing time: %f ms\n\n", time);
	total_time_sequential += time;

	timer.Start();
	apply_filter_(grayscalePixels, width, height, filter2, filterWidth, filteredPixels_2);
	timer.Stop();
	time = timer.Elapsed();
	printf("Apply y-Sobel filter - Processing time: %f ms\n\n", time);
	total_time_sequential += time;
	
	// Calculate importance of each pixel
	free(grayscalePixels); // Free grayscale matrix after done with it
	int * pixelImportance = (int *)malloc(width * height * sizeof(int));
	
	timer.Start();
	calc_px_importance(filteredPixels_1, filteredPixels_2, pixelImportance, width, height);
	timer.Stop();
	time = timer.Elapsed();
	printf("Calculate pixel importance - Processing time: %f ms\n\n", time);
	total_time_sequential += time;
	
	// Construct least pixel-importance matrix
	int * importantMatrix = (int *)malloc(width * height * sizeof(int));
	int * importantMatrixTrace = (int *)malloc(width * height * sizeof(int));

	timer.Start();
	create_important_matrix(pixelImportance, width, height, importantMatrix, importantMatrixTrace);
	timer.Stop();
	time = timer.Elapsed();
	printf("Construct least pixel-importance matrix - Processing time: %f ms\n\n", time);
	total_time_sequential += time;

	// Find K least important seams from the least pixel-importance matrix
	pair_int_int * k_best_list = (pair_int_int *)malloc(k * height * sizeof(pair_int_int));
	
	timer.Start();
	int actualK = get_k_best(importantMatrix, importantMatrixTrace, width, height, k, k_best_list);
	timer.Stop();
	time = timer.Elapsed();
	printf("Find K least important seams - Processing time: %f ms\n\n", time);
	total_time_sequential += time;
	printf("Needed %d seams. Actual seams found: %d\n\n", k, actualK);

	// Sort seam positions in each row for efficient remove/duplicate
	qsort(k_best_list, actualK * height, sizeof(pair_int_int), compare_position);

	// Remove or duplicate K seams to change image size
	uchar3 *outPixels = (uchar3 *)malloc(desiredWidth * height * sizeof(uchar3));

	timer.Start();
	applyKSeams(inPixels, outPixels, width, height, k_best_list, actualK, mode);
	timer.Stop();
	time = timer.Elapsed();
	printf("Reduce/Enlarge image - Processing time: %f ms\n\n", time);
	total_time_sequential += time;

	// Save output image
	writePnm(outPixels, 3, desiredWidth, height, outImg);

	// //############################################################
	// int * out_cuda = (int *)malloc(width * height * sizeof(int));
	// int * out_trace_cuda = (int *)malloc(width * height * sizeof(int));
	
	
	
	// get_k_best_cuda(important_matrix,important_matrix_trace, width, height, k, k_best_list,512);
	// for (int r = 0; r < k; r++) {
    //     for (int c = 0; c < height; c++)
	// 		printf("%i: %i %i \n",c, k_best_list[r*height + c].first, k_best_list[r*height+c].second);
	// 	printf("\n");
	// }

	// Write results to files
	// char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	// writePnm(applyKernelPixels, 1, width, height, concatStr(outFileNameBase, "_host.pnm"));

	// Output total run time
	printf("Total processing time: %f ms\n\n", total_time_sequential);

	// Free memories
	free(inPixels);
	free(filteredPixels_1);
	free(filteredPixels_2);
	free(pixelImportance);
	free(importantMatrix);
	free(importantMatrixTrace);
	free(k_best_list);
	free(outPixels);
}